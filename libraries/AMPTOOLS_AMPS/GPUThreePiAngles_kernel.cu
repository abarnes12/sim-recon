#include "hip/hip_runtime.h"
/*
 *  GPUThreePiAngles_kernel.cu
 *  GlueXTools
 *
 *  Created by Matthew Shepherd on 6/16/10.
 *  Copyright 2010 Home. All rights reserved.
 *
 */


#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"

#include "AMPTOOLS_AMPS/breakupMomentum.cuh"

#include "GPUUtils/lorentzBoost.cuh"
#include "GPUUtils/threeVector.cuh"
#include "GPUUtils/wignerD.cuh"
#include "GPUUtils/clebsch.cuh"

__global__ void
GPUThreePiAngles_kernel( GPU_AMP_PROTO, int polBeam, GDouble polFrac, int jX, 
                         int parX, int iX, int lX, int jI, int iI, int iZ0, 
                         int iZ1, int iZ2 ){

	int iEvent = GPU_THIS_EVENT;
  
  GDouble beam[4]   = GPU_P4(0);
  GDouble recoil[4] = GPU_P4(1);
	GDouble p1[4]     = GPU_P4(2);
	GDouble p2[4]     = GPU_P4(3);
  GDouble p3[4]     = GPU_P4(4);

  GDouble alpha = phi( &(recoil[1]) );

  GDouble res[4];
  GDouble iso[4];
  	
  for( int i = 0; i < 4; ++i ){ 
    
    iso[i] = p1[i] + p2[i];
    res[i] = iso[i] + p3[i];
  }
  
  GDouble resMass = G_SQRT(res[0]*res[0]-res[1]*res[1]-res[2]*res[2]-res[3]*res[3]);
  GDouble isoMass = G_SQRT(iso[0]*iso[0]-iso[1]*iso[1]-iso[2]*iso[2]-iso[3]*iso[3]);
  GDouble p1Mass = G_SQRT(p1[0]*p1[0]-p1[1]*p1[1]-p1[2]*p1[2]-p1[3]*p1[3]);
  GDouble p2Mass = G_SQRT(p2[0]*p2[0]-p2[1]*p2[1]-p2[2]*p2[2]-p2[3]*p2[3]);
  GDouble p3Mass = G_SQRT(p3[0]*p3[0]-p3[1]*p3[1]-p3[2]*p3[2]-p3[3]*p3[3]);

  GDouble k = breakupMomentum( resMass, isoMass, p3Mass );
  GDouble q = breakupMomentum( isoMass, p1Mass, p2Mass );

  boostToRest( beam   , res );
  boostToRest( recoil , res );
  boostToRest( iso    , res ); 
  boostToRest( p1     , res );

  // now beam, recoil, iso, and p1 are at rest in the resonance frame

  // create the z axis in this frame
  GDouble zRes[3] = { -recoil[1], -recoil[2], -recoil[3] };
  makeUnit( zRes );
  
  // create the y axis from the cross product of the beam with z
  GDouble yRes[3] = { beam[1], beam[2], beam[3] };
  cross( yRes, zRes );
  makeUnit( yRes );
  
  // defines x and replaces it with the cross product
  // of y and z
  GDouble xRes[3] = { yRes[0], yRes[1], yRes[2] };
  cross( xRes, zRes );

  // rewrite the isobar direction in this coordinate system
  GDouble angRes[3] = { dot( &(iso[1]), xRes ), 
                        dot( &(iso[1]), yRes ),
                        dot( &(iso[1]), zRes )  };

  // and record the angles
  GDouble cosThRes  = cosTheta( angRes );
  GDouble phiAngRes = phi( angRes );
  
  boostToRest( p1 , iso );
  
  GDouble angIso[3] = { dot( &(p1[1]), xRes ),
                        dot( &(p1[1]), yRes ),
                        dot( &(p1[1]), zRes ) };
                        
  GDouble cosThIso = cosTheta( angIso );
  GDouble phiAngIso = phi( angIso );

  WCUComplex i = { 0, 1 };
  WCUComplex one = { 1, 0 };
  WCUComplex ans = { 0, 0 };
  
  // a prefactor the matrix elements that couple negative helicity
  // photons to the final state
   WCUComplex negResHelProd = ( polBeam == 0 ? 
    ( one * G_COS( 2 * alpha ) + i * G_SIN( 2 * alpha ) ) :
    ( one * G_COS( 2 * alpha ) + i * G_SIN( 2 * alpha ) ) * -1 );
  negResHelProd *= ( jX % 2 == 0 ? -parX : parX );
 
  // in general we also need a sum over resonance helicities here
  // however, we assume a production mechanism that only produces
  // resonance helicities +-1
 
  for( int mL = -lX; mL <= lX; ++mL ){
    
    WCUComplex term = { 0, 0 };
    
    for( int mI = -jI; mI <= jI; ++mI ){
      
        // CAREFUL!! ordering of arguments for GPU routine clebsch
        // is different from CPU routine clebschGordan
                                
        term += Y( jI, mI, cosThIso, phiAngIso ) *
        ( negResHelProd * clebsch( jI, mI, lX, mL, jX, -1 ) + 
          clebsch( jI, mI, lX, mL, jX,  1 ) );
    }
    
    term *= Y( lX, mL, cosThRes, phiAngRes );
    ans += term;
  }
  
  ans *= ( polBeam == 0 ? ( 1 + polFrac ) / 4 : ( 1 - polFrac ) / 4 );
  
  pcDevAmp[iEvent] = ans * 
      clebsch( 1, iZ0, 1, iZ1, iI, iZ0 + iZ1 ) *
      clebsch( iI, iZ0 + iZ1, 1, iZ2, iX, iZ0 + iZ1 + iZ2 ) *
      G_POW( k, lX ) * G_POW( q, jI );
}

void
GPUThreePiAngles_exec( dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO,
                       int polBeam, GDouble polFrac, int jX, int parX, int iX, 
                       int lX, int jI, int iI, int iZ0, int iZ1, int iZ2 )  
{  
  GPUThreePiAngles_kernel<<< dimGrid, dimBlock >>>
     ( GPU_AMP_ARGS, polBeam, polFrac, jX, parX, iX, lX, jI, iI, iZ0, iZ1, iZ2 );
}

