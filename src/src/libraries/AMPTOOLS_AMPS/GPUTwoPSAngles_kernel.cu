#include "hip/hip_runtime.h"

#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"

#include "GPUUtils/lorentzBoost.cuh"
#include "GPUUtils/threeVector.cuh"
#include "GPUUtils/wignerD.cuh"

__global__ void
GPUTwoPSAngles_kernel( GPU_AMP_PROTO, int j, int m, GDouble bigTheta, 
                       GDouble refFact ){

	int iEvent = GPU_THIS_EVENT;
  
  GDouble beam[4]   = GPU_P4(0);
  GDouble recoil[4] = GPU_P4(1);
	GDouble p1[4]     = GPU_P4(2);
	GDouble p2[4]     = GPU_P4(3);

  GDouble res[4];
  	
  for( int i = 0; i < 4; ++i ) res[i] = p1[i] + p2[i];

  boostToRest( beam   , res );
  boostToRest( recoil , res );
  boostToRest( p1     , res ); 

  GDouble z[3] = { beam[1], beam[2], beam[3] };
  makeUnit( z );
  
  GDouble y[3] = { recoil[1], recoil[2], recoil[3] };
  cross( y, z );
  makeUnit( y );
  
  // defines x and replaces it with the cross product
  // of y and z
  GDouble x[3] = { y[0], y[1], y[2] };
  cross( x, z );

  GDouble ang[3] = { dot( &(p1[1]), x ), 
                     dot( &(p1[1]), y ),
                     dot( &(p1[1]), z )  };

  GDouble cosTh  = cosTheta( ang );
  GDouble phiAng = phi( ang );    
 
  GDouble coef   = sqrt( ( 2. * j + 1 ) / ( 4 * 3.1416 ) );
       
  pcDevAmp[iEvent] = 
     ( coef * bigTheta * 
        ( wignerD( j, m, 0, cosTh, phiAng ) - 
          refFact * wignerD( j, -m, 0, cosTh, phiAng ) ) );

}

void
GPUTwoPSAngles_exec( dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO,
                     int j, int m, GDouble bigTheta, GDouble refFact )  
{  
  GPUTwoPSAngles_kernel<<< dimGrid, dimBlock >>>
     ( GPU_AMP_ARGS, j, m, bigTheta, refFact );
}
