#include "hip/hip_runtime.h"

#include <stdio.h>

#include "GPUManager/GPUCustomTypes.h"
#include "GPUManager/CUDA-Complex.cuh"

#include "AMPTOOLS_AMPS/breakupMomentum.cuh"
#include "AMPTOOLS_AMPS/barrierFactor.cuh"

__global__ void
GPUBreitWigner_kernel( GPU_AMP_PROTO, GDouble mass0, GDouble width0, 
                       GDouble orbitL, int daught1, int daught2 ){

	int iEvent = GPU_THIS_EVENT;

  // decode the list of daughter indices in each integer
  // be careful to handle the zero index correctly

  GDouble p1[4] = { 0, 0, 0, 0 };
  if( daught1 == 0 ){
  
    GDouble tmp[4] = GPU_P4(0);
    for( int i = 0; i < 4; ++i ) p1[i] += tmp[i];
  } 
  else if( daught1 > 0 ){
  
    while( daught1 > 0 ){
    
      int ind = daught1 % 10;
      GDouble tmp[4] = GPU_P4(ind);
      for( int i = 0; i < 4; ++i ) p1[i] += tmp[i];
      daught1 /= 10;
    }
  }
  
  
  GDouble p2[4] = { 0, 0, 0, 0 };
  if( daught2 == 0 ){
  
    GDouble tmp[4] = GPU_P4(0);
    for( int i = 0; i < 4; ++i ) p2[i] += tmp[i];
  } 
  else if( daught2 > 0 ){
  
    while( daught2 > 0 ){
    
      int ind = daught2 % 10;
      GDouble tmp[4] = GPU_P4(ind);
      for( int i = 0; i < 4; ++i ) p2[i] += tmp[i];
      daught2 /= 10;
    }
  }


  GDouble mass  = SQ( p1[0] + p2[0] );
  GDouble mass1 = SQ( p1[0] );
  GDouble mass2 = SQ( p2[0] );
   
  for( int i = 1; i <= 3; ++i ){
    
    mass  -= SQ( p1[i] + p2[i] );
    mass1 -= SQ( p1[i] );
    mass2 -= SQ( p2[i] );
  }
  
  mass  = G_SQRT( mass  );
  mass1 = G_SQRT( mass1 );
  mass2 = G_SQRT( mass2 );

  GDouble q  = fabs( breakupMomentum(  mass, mass1, mass2 ) );
  GDouble q0 = fabs( breakupMomentum( mass0, mass1, mass2 ) );

  GDouble F  = barrierFactor( q,  orbitL );
  GDouble F0 = barrierFactor( q0, orbitL );
  
  GDouble width = width0*(mass0/mass)*(q/q0)*((F*F)/(F0*F0));
//  GDouble width = width0;
 
  WCUComplex bwTop = { G_SQRT( mass0 * width0 / 3.1416 ), 0 };
  WCUComplex bwBot = { SQ( mass0 ) - SQ( mass ), -1.0 * mass0 * width };

  pcDevAmp[iEvent] = ( F * bwTop / bwBot );
}


void
GPUBreitWigner_exec( dim3 dimGrid, dim3 dimBlock, GPU_AMP_PROTO, 
                     GDouble mass, GDouble width, int orbitL,
                     int daught1, int daught2 )
{  

  GPUBreitWigner_kernel<<< dimGrid, dimBlock >>>
    ( GPU_AMP_ARGS, mass, width, orbitL, daught1, daught2 );
}
